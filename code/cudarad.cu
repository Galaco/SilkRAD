#include "hip/hip_runtime.h"
#include "cub/device/device_scan.cuh"

#include "hip/hip_runtime.h"
#include ""
#include "device_atomic_functions.h"

#include <iostream>
#include <thread>
#include <chrono>

#include "cudarad.h"

#include "bsp.h"

#include "cudabsp.h"
#include "cudamatrix.h"
#include "raytracer.h"

#include "cudautils.h"

#define MAX_ITER 100
#define PRECOMPUTE false


namespace CUDARAD {
    static std::unique_ptr<RayTracer::CUDARayTracer> g_pRayTracer;
    static __device__ RayTracer::CUDARayTracer* g_pDeviceRayTracer;

    __device__ FaceInfo::FaceInfo() {};

    __device__ FaceInfo::FaceInfo(
            CUDABSP::CUDABSP& cudaBSP,
            size_t faceIndex
            ) :
            faceIndex(faceIndex),
            face(cudaBSP.faces[faceIndex]),
            plane(cudaBSP.planes[face.planeNum]),
            texInfo(cudaBSP.texInfos[face.texInfo]),
            Ainv(cudaBSP.xyzMatrices[faceIndex]),
            faceNorm(
                make_float3(plane.normal.x, plane.normal.y, plane.normal.z)
            ),
            lightmapWidth(face.lightmapTextureSizeInLuxels[0] + 1),
            lightmapHeight(face.lightmapTextureSizeInLuxels[1] + 1),
            lightmapSize(lightmapWidth * lightmapHeight),
            lightmapStartIndex(face.lightOffset / sizeof(BSP::RGBExp32)),
            totalLight(make_float3(0.0, 0.0, 0.0)) {}

    __device__ float3 FaceInfo::xyz_from_st(float s, float t) {
        float sOffset = this->texInfo.lightmapVecs[0][3];
        float tOffset = this->texInfo.lightmapVecs[1][3];

        float sMin = this->face.lightmapTextureMinsInLuxels[0];
        float tMin = this->face.lightmapTextureMinsInLuxels[1];

        CUDAMatrix::CUDAMatrix<double, 3, 1> B;

        B[0][0] = s - sOffset + sMin;
        B[1][0] = t - tOffset + tMin;
        B[2][0] = this->plane.dist;

        CUDAMatrix::CUDAMatrix<double, 3, 1> result = this->Ainv * B;

        return make_float3(result[0][0], result[1][0], result[2][0]);
    }
}


namespace DirectLighting {
    static __device__ inline float attenuate(
            BSP::DWorldLight& light,
            float dist
            ) {

        float c = light.constantAtten;
        float l = light.linearAtten;
        float q = light.quadraticAtten;

        return c + l * dist + q * dist * dist;
    }

    __device__ float3 sample_at(
            CUDABSP::CUDABSP& cudaBSP,
            float3 samplePos,
            float3 sampleNormal=make_float3(0.0, 0.0, 0.0)
            ) {

        //samplePos += faceInfo.faceNorm * 1e-3;

        float3 result = make_float3(0.0, 0.0, 0.0);

        for (size_t lightIndex=0;
                 lightIndex<cudaBSP.numWorldLights;
                 lightIndex++
                 ) {

            BSP::DWorldLight& light = cudaBSP.worldLights[lightIndex];

            float3 lightPos = make_float3(
                light.origin.x,
                light.origin.y,
                light.origin.z
            );

            float3 diff = samplePos - lightPos;

            /*
             * This light is on the wrong side of the current sample.
             * There's no way it could possibly light it.
             */
            if (len(sampleNormal) > 0.0 && dot(diff, sampleNormal) >= 0.0) {
                continue;
            }

            float dist = len(diff);
            float3 dir = diff / dist;

            float penumbraScale = 1.0;

            if (light.type == BSP::EMIT_SPOTLIGHT) {
                float3 lightNorm = make_float3(
                    light.normal.x,
                    light.normal.y,
                    light.normal.z
                );

                float lightDot = dot(dir, lightNorm);

                if (lightDot < light.stopdot2) {
                    /* This sample is outside the spotlight cone. */
                    continue;
                }
                else if (lightDot < light.stopdot) {
                    /* This sample is within the spotlight's penumbra. */
                    penumbraScale = (
                        (lightDot - light.stopdot2)
                        / (light.stopdot - light.stopdot2)
                    );
                    //penumbraScale = 100.0;
                }

                //if (lightIndex == cudaBSP.numWorldLights - 1) {
                //    printf(
                //        "(%f, %f, %f) is within spotlight!\n"
                //        "Pos: (%f, %f, %f)\n"
                //        "Norm: <%f, %f, %f> (<%f, %f, %f>)\n"
                //        "stopdot: %f; stopdot2: %f\n"
                //        "Dot between light and sample: %f\n",
                //        samplePos.x, samplePos.y, samplePos.z,
                //        lightPos.x, lightPos.y, lightPos.z,
                //        lightNorm.x, lightNorm.y, lightNorm.z,
                //        light.normal.x, light.normal.y, light.normal.z,
                //        light.stopdot, light.stopdot2,
                //        lightDot
                //    );
                //}
            }

            const float EPSILON = 1e-3;

            // Nudge the sample position towards the light slightly, to avoid
            // colliding with triangles that directly contain the sample
            // position.
            samplePos -= dir * EPSILON;

            bool lightBlocked = CUDARAD::g_pDeviceRayTracer->LOS_blocked(
                lightPos, samplePos
            );

            if (lightBlocked) {
                // This light can't be seen from the position of the sample.
                // Ignore it.
                continue;
            }

            /* I CAN SEE THE LIGHT */
            float attenuation = attenuate(light, dist);

            float3 lightContribution = make_float3(
                light.intensity.x,  // r
                light.intensity.y,  // g
                light.intensity.z   // b
            );

            lightContribution *= penumbraScale * 255.0 / attenuation;

            result += lightContribution;
        }

        //printf(
        //    "Sample at (%u, %u) for Face %u: (%f, %f, %f)\n",
        //    static_cast<unsigned int>(s),
        //    static_cast<unsigned int>(t),
        //    static_cast<unsigned int>(faceIndex),
        //    result.x, result.y, result.z
        //);

        return result;
    }

    __device__ float3 sample_at(
            CUDABSP::CUDABSP& cudaBSP,
            CUDARAD::FaceInfo& faceInfo,
            float s, float t
            ) {

        float3 samplePos = faceInfo.xyz_from_st(s, t);
        return sample_at(cudaBSP, samplePos, faceInfo.faceNorm);
    }

    __global__ void map_faces(
            CUDABSP::CUDABSP* pCudaBSP,
            size_t* pFacesCompleted
            ) {

        bool primaryThread = (threadIdx.x == 0 && threadIdx.y == 0);

        if (pCudaBSP->tag != CUDABSP::TAG) {
            if (primaryThread) {
                printf("Invalid CUDABSP Tag: %x\n", pCudaBSP->tag);
            }
            return;
        }

        __shared__ CUDARAD::FaceInfo faceInfo;

        if (primaryThread) {
            // Map block numbers to faces.
            faceInfo = CUDARAD::FaceInfo(*pCudaBSP, blockIdx.x);

            //printf(
            //    "Processing Face %u...\n",
            //    static_cast<unsigned int>(faceInfo.faceIndex)
            //);
        }

        __syncthreads();

        /* Take a sample at each lightmap luxel. */
        for (size_t i=0; i<faceInfo.lightmapHeight; i+=blockDim.y) {
            size_t t = i + threadIdx.y;

            if (t >= faceInfo.lightmapHeight) {
                continue;
            }

            for (size_t j=0; j<faceInfo.lightmapWidth; j+=blockDim.x) {
                size_t s = j + threadIdx.x;

                if (s >= faceInfo.lightmapWidth) {
                    continue;
                }

                float3 color = sample_at(
                    *pCudaBSP, faceInfo,
                    static_cast<float>(s),
                    static_cast<float>(t)
                );

                size_t& lightmapStart = faceInfo.lightmapStartIndex;
                size_t sampleIndex = t * faceInfo.lightmapWidth + s;

                pCudaBSP->lightSamples[lightmapStart + sampleIndex] = color;

                atomicAdd(&faceInfo.totalLight.x, color.x);
                atomicAdd(&faceInfo.totalLight.y, color.y);
                atomicAdd(&faceInfo.totalLight.z, color.z);
            }
        }

        __syncthreads();

        if (primaryThread) {
            faceInfo.avgLight = faceInfo.totalLight;
            faceInfo.avgLight /= static_cast<float>(faceInfo.lightmapSize);

            pCudaBSP->lightSamples[faceInfo.lightmapStartIndex - 1]
                = faceInfo.avgLight;

            // Still have no idea how this works. But if we don't do this,
            // EVERYTHING becomes a disaster...
            faceInfo.face.styles[0] = 0x00;
            faceInfo.face.styles[1] = 0xFF;
            faceInfo.face.styles[2] = 0xFF;
            faceInfo.face.styles[3] = 0xFF;

            /* Copy our changes back to the CUDABSP. */
            pCudaBSP->faces[faceInfo.faceIndex] = faceInfo.face;

            atomicAdd(reinterpret_cast<unsigned int*>(pFacesCompleted), 1);
            __threadfence_system();
        }

        //printf(
        //    "Lightmap offset for face %u: %u\n",
        //    static_cast<unsigned int>(faceIndex),
        //    static_cast<unsigned int>(lightmapStartIndex)
        //);

        //printf("%u\n", static_cast<unsigned int>(*pFacesCompleted));
    }
}


namespace AA {
    static __device__ const float INV_GAMMA = 1.0 / 2.2;

    static __device__ inline float perceptual_from_linear(float linear) {
        return powf(linear, INV_GAMMA);
    }

    static __device__ float intensity(float3 rgb) {
        return perceptual_from_linear(
            dot(
                rgb / 255.0,
                make_float3(1.0, 1.0, 1.0)
                //make_float3(0.299, 0.587, 0.114)
            )
        );
    }

    static __device__ const float MIN_AA_GRADIENT = 0.125;      // 1/8
    //static __device__ const float MIN_AA_GRADIENT = 0.0625;    // 1/16

    __global__ void map_face_samples(
            CUDABSP::CUDABSP* pCudaBSP,
            /* output */ int* facesForSamples,
            /* output */ int2* coordsForSamples
            ) {

        int faceIndex = blockIdx.x * blockDim.x + threadIdx.x;

        if (faceIndex >= pCudaBSP->numFaces) {
            return;
        }

        CUDARAD::FaceInfo faceInfo(*pCudaBSP, faceIndex);

        for (size_t i=0; i<faceInfo.lightmapSize; i++) {
            size_t sampleIndex = faceInfo.lightmapStartIndex + i;

            facesForSamples[sampleIndex] = faceIndex;
            coordsForSamples[sampleIndex] = make_int2(
                i % faceInfo.lightmapWidth,
                i / faceInfo.lightmapWidth
            );
        }
    }

    __global__ void map_select_targets(
            CUDABSP::CUDABSP* pCudaBSP,
            int* facesForSamples, int2* coordsForSamples,
            /* output */ int* targets
            ) {

        size_t sampleIndex = blockIdx.x * blockDim.x + threadIdx.x;

        if (sampleIndex >= pCudaBSP->numLightSamples) {
            return;
        }

        int faceIndex = facesForSamples[sampleIndex];
        int2 coords = coordsForSamples[sampleIndex];

        if (faceIndex == -1 || coords.x == -1 || coords.y == -1) {
            return;
        }

        int s = coords.x;
        int t = coords.y;

        CUDARAD::FaceInfo faceInfo(*pCudaBSP, faceIndex);

        float3* samples = &pCudaBSP->lightSamples[faceInfo.lightmapStartIndex];

        size_t width = faceInfo.lightmapWidth;
        size_t height = faceInfo.lightmapHeight;

        float3 sample = samples[t * width + s];

        float gradient = 0.0;

        for (int offsetT=-1; offsetT<=1; offsetT++) {
            int neighborT = t + offsetT;

            if (!(0 <= neighborT && neighborT < height)) {
                continue;
            }

            for (int offsetS=-1; offsetS<=1; offsetS++) {
                if (offsetS == 0 && offsetT == 0) {
                    continue;
                }

                int neighborS = s + offsetS;

                if (!(0 <= neighborS && neighborS < width)) {
                    continue;
                }

                size_t neighborIndex = neighborT * width + neighborS;

                float3 neighbor = samples[neighborIndex];

                gradient = fmaxf(
                    gradient,
                    fabsf(intensity(neighbor) - intensity(sample))
                );
            }
        }

        targets[sampleIndex] = static_cast<int>(gradient >= MIN_AA_GRADIENT);
    }

    __global__ void gather_target_coords(
            CUDABSP::CUDABSP* pCudaBSP,
            /* output */ int2* finalCoords,
            /* output */ int* finalFacesForCoords,
            int2* coordsForSamples, int* facesForSamples,
            int* targetsScanned, int* targets
            ) {

        size_t index = blockIdx.x * blockDim.x + threadIdx.x;

        if (index >= pCudaBSP->numLightSamples) {
            return;
        }

        if (targets[index]) {
            size_t destIndex = targetsScanned[index] - 1;

            finalCoords[destIndex] = coordsForSamples[index];
            finalFacesForCoords[destIndex] = facesForSamples[index];
        }
    }

    __global__ void antialias_coords(
            CUDABSP::CUDABSP* pCudaBSP,
            int2* coords, int* facesForCoords,
            size_t numCoords
            ) {

        size_t index = blockIdx.x * blockDim.x + threadIdx.x;

        if (index >= numCoords) {
            return;
        }

        int2 samplePos = coords[index];
        size_t faceIndex = facesForCoords[index];

        CUDARAD::FaceInfo faceInfo(*pCudaBSP, faceIndex);

        int s = samplePos.x;
        int t = samplePos.y;

        /* Perform supersampling at this point. */
        const size_t SUPERSAMPLE_WIDTH = 4;

        float sStep = 2.0 / static_cast<float>(SUPERSAMPLE_WIDTH);
        float tStep = 2.0 / static_cast<float>(SUPERSAMPLE_WIDTH);

        float3 color = make_float3(0.0, 0.0, 0.0);

        for (size_t ssi=0; ssi<SUPERSAMPLE_WIDTH; ssi++) {
            float tOffset = tStep * ssi - 1.0;

            for (size_t ssj=0; ssj<SUPERSAMPLE_WIDTH; ssj++) {
                float sOffset = sStep * ssj - 1.0;

                color += DirectLighting::sample_at(
                    *pCudaBSP, faceInfo,
                    s + sOffset, t + tOffset
                );
            }
        }

        color /= SUPERSAMPLE_WIDTH * SUPERSAMPLE_WIDTH;

        size_t startIndex = faceInfo.lightmapStartIndex;
        size_t sampleIndex = t * faceInfo.lightmapWidth + s;

        pCudaBSP->lightSamples[startIndex + sampleIndex] = color;
    }

    __global__ void map_faces_AA(CUDABSP::CUDABSP* pCudaBSP) {
        bool primaryThread = (threadIdx.x == 0 && threadIdx.y == 0);

        __shared__ CUDARAD::FaceInfo faceInfo;

        __shared__ size_t lightmapStart;
        __shared__ size_t width;
        __shared__ size_t height;

        //__shared__ float3* results;

        if (primaryThread) {
            // Map block numbers to faces.
            faceInfo = CUDARAD::FaceInfo(*pCudaBSP, blockIdx.x);

            lightmapStart = faceInfo.lightmapStartIndex;
            width = faceInfo.lightmapWidth;
            height = faceInfo.lightmapHeight;

            //results = new float3[width * height];
        }

        __syncthreads();

        for (size_t i=0; i<height; i+=blockDim.y) {
            size_t t = i + threadIdx.y;

            if (t >= height) {
                continue;
            }

            for (size_t j=0; j<width; j+=blockDim.x) {
                size_t s = j + threadIdx.x;

                if (s >= width) {
                    continue;
                }

                size_t sampleIndex = t * width + s;

                float3 sampleColor
                    = pCudaBSP->lightSamples[lightmapStart + sampleIndex];

                float sampleIntensity = intensity(sampleColor);

                /* Calculate the maximum gradient of this luxel. */
                float gradient = 0.0;

                for (int tOffset=-1; tOffset<=1; tOffset++) {
                    int neighborT = t + tOffset;

                    if (!(0 <= neighborT && neighborT < height)) {
                        continue;
                    }

                    for (int sOffset=-1; sOffset<=1; sOffset++) {
                        if (sOffset == 0 && tOffset == 0) {
                            continue;
                        }

                        int neighborS = s + sOffset;

                        if (!(0 <= neighborS && neighborS < width)) {
                            continue;
                        }

                        int neighborIndex
                            = neighborT * width + neighborS;

                        float neighborIntensity = intensity(
                            pCudaBSP->lightSamples[
                                lightmapStart + neighborIndex
                            ]
                        );

                        gradient = fmaxf(
                            gradient,
                            fabsf(neighborIntensity - sampleIntensity)
                        );
                    }
                }

                /*
                * Don't bother antialiasing this sample if the gradient is
                * low enough.
                */
                if (gradient < MIN_AA_GRADIENT) {
                    //results[sampleIndex] = sampleColor;
                    continue;
                }

                ///*
                // * Box blur!
                // * Really stupid and potentially ugly, but really fast!
                // */

                //float3 color = make_float3(0.0, 0.0, 0.0);

                //for (int tOffset=-1; tOffset<=1; tOffset++) {
                //    float blurT = static_cast<float>(t) + tOffset;

                //    for (int sOffset=-1; sOffset<=1; sOffset++) {
                //        float blurS = static_cast<float>(s) + sOffset;

                //        float3 blurColor;

                //        /*
                //         * Out of range!
                //         * We have no choice but to actually take a sample.
                //         */
                //        if (!(0 <= blurS && blurS < width)
                //                || !(0 <= blurT && blurT < height)) {
                //            blurColor = sample_at(
                //                *pCudaBSP, faceInfo,
                //                blurS, blurT
                //            );
                //        }
                //        else {
                //            size_t i = static_cast<size_t>(
                //                blurT * width + blurS
                //            );
                //            blurColor = pCudaBSP->lightSamples[
                //                lightmapStart + i
                //            ];
                //        }

                //        color += blurColor;
                //    }
                //}

                ///* Take the average of the box blur samples. */
                //color /= 9.0;

                /* Perform supersampling at this point. */
                const size_t SUPERSAMPLE_WIDTH = 4;

                float sStep = 2.0 / static_cast<float>(SUPERSAMPLE_WIDTH);
                float tStep = 2.0 / static_cast<float>(SUPERSAMPLE_WIDTH);

                float3 color = make_float3(0.0, 0.0, 0.0);

                for (size_t ssi=0; ssi<SUPERSAMPLE_WIDTH; ssi++) {
                    float tOffset = tStep * ssi - 1.0;

                    for (size_t ssj=0; ssj<SUPERSAMPLE_WIDTH; ssj++) {
                        float sOffset = sStep * ssj - 1.0;

                        color += DirectLighting::sample_at(
                            *pCudaBSP, faceInfo,
                            s + sOffset, t + tOffset
                        );
                    }
                }

                color /= SUPERSAMPLE_WIDTH * SUPERSAMPLE_WIDTH;

                //results[sampleIndex] = color;
                pCudaBSP->lightSamples[lightmapStart + sampleIndex] = color;
            }
        }

        //__syncthreads();

        //if (primaryThread) {
        //    /* Move the results back to the light samples array. */
        //    memcpy(
        //        pCudaBSP->lightSamples + faceInfo.lightmapStartIndex,
        //        results,
        //        sizeof(float3) * faceInfo.lightmapSize
        //    );

        //    delete[] results;
        //}
    }
}


namespace BouncedLighting {
    static __device__ const float PI = 3.14159265358979323846264;
    static __device__ const float INV_PI = 0.31830988618379067153715;


    /**
     * Computes the form factor from a differential patch to a convex
     * polygonal patch.
     *
     * Thankfully, Source's polygons are always convex.
     *
     * Formula graciously stolen from Formula 81 of this book:
     * https://people.cs.kuleuven.be/~philip.dutre/GI/TotalCompendium.pdf
     *
     * ... and Formula 4.16 of this one:
     * https://books.google.com/books?id=zALK286TFXgC&lpg=PP1&pg=PA72#v=onepage&q&f=false
     */
    static __device__ float ff_diff_poly(
            float3 diffPos, float3 diffNorm,
            float3* vertices, size_t numVertices
            ) {

        float result = 0.0;

        for (size_t i=0; i<4; i++) {
            float3 vertex1 = vertices[i] - diffPos;
            float3 vertex2 = vertices[(i + 1) % numVertices] - diffPos;
            float3 vertexCross = cross(vertex1, vertex2);
            float crossLen = len(vertexCross);

            vertexCross /= crossLen;

            float v1Len = len(vertex1);
            float v2Len = len(vertex2);

            float theta =  asinf(crossLen / (v1Len * v2Len));

            result += dot(diffNorm, vertexCross) * theta;
        }

        result *= 0.5 * INV_PI;

        return result;
    }


    /** Computes the form factor between two differential patches. */
    static __device__ float ff_diff_diff(
            float3 diff1Pos, float3 diff1Norm,
            float3 diff2Pos, float3 diff2Norm
            ) {

        float3 delta = diff2Pos - diff1Pos;
        float invDist = 1.0 / len(delta);

        float3 dir = delta * invDist;

        return (
            dot(diff1Norm, dir) * -dot(diff2Norm, dir)
            * INV_PI * invDist * invDist
        );
    }
}


namespace AmbientLighting {
    static __device__ const float AMBIENT_SCALE = 0.0078125;    // 1/128

    __global__ void map_leaves(CUDABSP::CUDABSP* pCudaBSP) {
        size_t leafIndex = blockIdx.x;

        if (leafIndex >= pCudaBSP->numLeaves) {
            return;
        }

        BSP::DLeaf& leaf = pCudaBSP->leaves[leafIndex];

        if (leaf.contents & BSP::CONTENTS_SOLID) {
            return;
        }

        BSP::DLeafAmbientIndex& ambientIndex
            = pCudaBSP->ambientIndices[leafIndex];

        BSP::DLeafAmbientLighting* ambientSamples
            = &pCudaBSP->ambientLightSamples[ambientIndex.firstAmbientSample];

        for (size_t i=threadIdx.x;
                i<ambientIndex.ambientSampleCount;
                i+=blockDim.x) {

            if (i >= ambientIndex.ambientSampleCount) {
                return;
            }

            BSP::DLeafAmbientLighting& sample = ambientSamples[i];

            float3 leafMins = make_float3(
                leaf.mins[0], leaf.mins[1], leaf.mins[2]
            );

            float3 leafMaxs = make_float3(
                leaf.maxs[0], leaf.maxs[1], leaf.maxs[2]
            );

            float3 leafSize = leafMaxs - leafMins;

            float3 samplePos = leafMins + make_float3(
                leafSize.x * static_cast<float>(sample.x) / 255.0,
                leafSize.y * static_cast<float>(sample.y) / 255.0,
                leafSize.z * static_cast<float>(sample.z) / 255.0
            );

            //sample.cube.color[0] = BSP::RGBExp32 {1, 1, 1, -3};
            //sample.cube.color[1] = BSP::RGBExp32 {1, 1, 1, -3};
            //sample.cube.color[2] = BSP::RGBExp32 {1, 1, 1, -3};
            //sample.cube.color[3] = BSP::RGBExp32 {1, 1, 1, -3};
            //sample.cube.color[4] = BSP::RGBExp32 {1, 1, 1, -3};
            //sample.cube.color[5] = BSP::RGBExp32 {1, 1, 1, -3};

            // +X
            sample.cube.color[0] = CUDABSP::rgbexp32_from_float3(
                DirectLighting::sample_at(
                    *pCudaBSP,
                    samplePos,
                    make_float3(1.0, 0.0, 0.0)
                ) * AMBIENT_SCALE
            );

            // -X
            sample.cube.color[1] = CUDABSP::rgbexp32_from_float3(
                DirectLighting::sample_at(
                    *pCudaBSP,
                    samplePos,
                    make_float3(-1.0, 0.0, 0.0)
                ) * AMBIENT_SCALE
            );

            // +Y
            sample.cube.color[2] = CUDABSP::rgbexp32_from_float3(
                DirectLighting::sample_at(
                    *pCudaBSP,
                    samplePos,
                    make_float3(0.0, 1.0, 0.0)
                ) * AMBIENT_SCALE
            );

            // -Y
            sample.cube.color[3] = CUDABSP::rgbexp32_from_float3(
                DirectLighting::sample_at(
                    *pCudaBSP,
                    samplePos,
                    make_float3(0.0, -1.0, 0.0)
                ) * AMBIENT_SCALE
            );

            // +Z
            sample.cube.color[4] = CUDABSP::rgbexp32_from_float3(
                DirectLighting::sample_at(
                    *pCudaBSP,
                    samplePos,
                    make_float3(0.0, 0.0, 1.0)
                ) * AMBIENT_SCALE
            );

            // -Z
            sample.cube.color[5] = CUDABSP::rgbexp32_from_float3(
                DirectLighting::sample_at(
                    *pCudaBSP,
                    samplePos,
                    make_float3(0.0, 0.0, -1.0)
                ) * AMBIENT_SCALE
            );
        }
    }
}


namespace CUDARAD {
    void init(BSP::BSP& bsp) {
        std::cout << "Setting up ray-trace acceleration structure... "
            << std::flush;

        using Clock = std::chrono::high_resolution_clock;

        auto start = Clock::now();

        g_pRayTracer = std::unique_ptr<RayTracer::CUDARayTracer>(
            new RayTracer::CUDARayTracer()
        );

        std::vector<RayTracer::Triangle> triangles;

        /* Put all of the BSP's face triangles into the ray-tracer. */
        for (const BSP::Face& face : bsp.get_faces()) {
            int32_t flags = face.get_texinfo().flags;

            if ((flags & BSP::SURF_TRANS) && !(flags & BSP::SURF_NODRAW)) {
                // Skip translucent faces, but keep nodraw faces.
                continue;
            }

            std::vector<BSP::Edge>::const_iterator pEdge
                = face.get_edges().begin();

            BSP::Vec3<float> vertex1 = (pEdge++)->vertex1;
            BSP::Vec3<float> vertex2;
            BSP::Vec3<float> vertex3 = (pEdge++)->vertex1;

            do {
                vertex2 = vertex3;
                vertex3 = (pEdge++)->vertex1;

                RayTracer::Triangle tri {
                    {
                        make_float3(vertex1.x, vertex1.y, vertex1.z),
                        make_float3(vertex2.x, vertex2.y, vertex2.z),
                        make_float3(vertex3.x, vertex3.y, vertex3.z),
                    },
                };

                triangles.push_back(tri);

            } while (pEdge != face.get_edges().end());
        }

        g_pRayTracer->add_triangles(triangles);

        auto end = Clock::now();
        std::chrono::milliseconds ms
            = std::chrono::duration_cast<std::chrono::milliseconds>(
                end - start
            );

        std::cout << "Done! (" << ms.count() << " ms)" << std::endl;

        std::cout << "Moving ray-tracer to device..." << std::endl;

        RayTracer::CUDARayTracer* pDeviceRayTracer;

        CUDA_CHECK_ERROR(
            hipMalloc(&pDeviceRayTracer, sizeof(RayTracer::CUDARayTracer))
        );
        CUDA_CHECK_ERROR(
            hipMemcpy(
                pDeviceRayTracer, g_pRayTracer.get(),
                sizeof(RayTracer::CUDARayTracer),
                hipMemcpyHostToDevice
            )
        );
        CUDA_CHECK_ERROR(
            hipMemcpyToSymbol(HIP_SYMBOL(
                g_pDeviceRayTracer), &pDeviceRayTracer,
                sizeof(RayTracer::CUDARayTracer*), 0,
                hipMemcpyHostToDevice
            )
        );
    }

    void cleanup(void) {
        RayTracer::CUDARayTracer* pDeviceRayTracer;

        CUDA_CHECK_ERROR(
            hipMemcpyFromSymbol(
                &pDeviceRayTracer, HIP_SYMBOL(g_pDeviceRayTracer),
                sizeof(RayTracer::CUDARayTracer*), 0,
                hipMemcpyDeviceToHost
            )
        );

        CUDA_CHECK_ERROR(hipFree(pDeviceRayTracer));

        g_pRayTracer = nullptr;
    }

    void compute_direct_lighting(BSP::BSP& bsp, CUDABSP::CUDABSP* pCudaBSP) {
        volatile size_t* pFacesCompleted;
        CUDA_CHECK_ERROR(
            hipHostAlloc(
                &pFacesCompleted, sizeof(size_t),
                hipHostMallocMapped
            )
        );

        *pFacesCompleted = 0;

        volatile size_t* pDeviceFacesCompleted;
        CUDA_CHECK_ERROR(
            hipHostGetDevicePointer(
                const_cast<size_t**>(&pDeviceFacesCompleted),
                const_cast<size_t*>(pFacesCompleted),
                0
            )
        );

        const size_t BLOCK_WIDTH = 16;
        const size_t BLOCK_HEIGHT = 16;

        size_t numFaces = bsp.get_faces().size();

        dim3 blockDim(BLOCK_WIDTH, BLOCK_HEIGHT);

        std::cout << "Launching "
            << numFaces * BLOCK_WIDTH * BLOCK_HEIGHT << " threads ("
            << numFaces << " faces)..."
            << std::endl;

        hipEvent_t startEvent;
        hipEvent_t stopEvent;

        CUDA_CHECK_ERROR(hipEventCreate(&startEvent));
        CUDA_CHECK_ERROR(hipEventCreate(&stopEvent));

        CUDA_CHECK_ERROR(hipEventRecord(startEvent));

        KERNEL_LAUNCH(
            DirectLighting::map_faces,
            numFaces, blockDim,
            pCudaBSP, const_cast<size_t*>(pDeviceFacesCompleted)
        );

        flush_wddm_queue();

        size_t lastFacesCompleted = 0;
        size_t facesCompleted;

        /* Progress notification logic */
        do {
            CUDA_CHECK_ERROR(hipPeekAtLastError());

            facesCompleted = *pFacesCompleted;

            if (facesCompleted > lastFacesCompleted) {
                std::cout << "    " << facesCompleted << "/"
                    << numFaces
                    << " faces processed..." << std::endl;
            }

            lastFacesCompleted = facesCompleted;

            std::this_thread::sleep_for(std::chrono::milliseconds(5));

        } while (facesCompleted < numFaces);

        CUDA_CHECK_ERROR(hipEventRecord(stopEvent));
        CUDA_CHECK_ERROR(hipDeviceSynchronize());

        float time;
        CUDA_CHECK_ERROR(hipEventElapsedTime(&time, startEvent, stopEvent));

        std::cout << "Done! (" << time << " ms)" << std::endl;

        hipHostFree(const_cast<size_t*>(pFacesCompleted));
    }

    void antialias_direct_lighting(BSP::BSP& bsp, CUDABSP::CUDABSP* pCudaBSP) {
        hipEvent_t startEvent;
        hipEvent_t stopEvent;

        CUDA_CHECK_ERROR(hipEventCreate(&startEvent));
        CUDA_CHECK_ERROR(hipEventCreate(&stopEvent));

        CUDA_CHECK_ERROR(hipEventRecord(startEvent));

        size_t numSamples = bsp.get_lightsamples().size();

        int* facesForSamples;

        CUDA_CHECK_ERROR(
            hipMalloc(&facesForSamples, sizeof(int) * numSamples)
        );
        CUDA_CHECK_ERROR(
            hipMemset(facesForSamples, -1, sizeof(int) * numSamples)
        );

        int2* coordsForSamples;

        CUDA_CHECK_ERROR(
            hipMalloc(&coordsForSamples, sizeof(int2) * numSamples)
        );
        CUDA_CHECK_ERROR(
            hipMemset(coordsForSamples, -1, sizeof(int2) * numSamples)
        );

        int* targets;
        CUDA_CHECK_ERROR(hipMalloc(&targets, sizeof(int) * numSamples));
        CUDA_CHECK_ERROR(hipMemset(targets, 0, sizeof(int) * numSamples));

        size_t blockWidth = 1024;
        size_t numBlocks = div_ceil(bsp.get_faces().size(), blockWidth);

        KERNEL_LAUNCH(
            AA::map_face_samples,
            numBlocks, blockWidth,
            pCudaBSP,
            facesForSamples, coordsForSamples
        );

        numBlocks = div_ceil(numSamples, blockWidth);

        KERNEL_LAUNCH(
            AA::map_select_targets,
            numBlocks, blockWidth,
            pCudaBSP,
            facesForSamples, coordsForSamples,
            targets
        );

        CUDA_CHECK_ERROR(hipDeviceSynchronize());

        int* targetsScanned;
        CUDA_CHECK_ERROR(
            hipMalloc(&targetsScanned, sizeof(int) * numSamples)
        );

        void* dTempStorage = nullptr;
        size_t tempStorageSize = 0;
        hipcub::DeviceScan::InclusiveSum(
            dTempStorage, tempStorageSize,
            targets, targetsScanned,
            numSamples
        );

        CUDA_CHECK_ERROR(hipMalloc(&dTempStorage, tempStorageSize));

        hipcub::DeviceScan::InclusiveSum(
            dTempStorage, tempStorageSize,
            targets, targetsScanned,
            numSamples
        );

        CUDA_CHECK_ERROR(hipFree(dTempStorage));

        int numTargets;
        CUDA_CHECK_ERROR(
            hipMemcpy(
                &numTargets, &targetsScanned[numSamples - 1], sizeof(int),
                hipMemcpyDeviceToHost
            )
        );

        if (numTargets <= 0) {
            // TODO: Free all hipMalloc()'d memory!
            return;
        }

        std::cout << "numTargets: " << numTargets << std::endl;

        int2* finalCoords;
        CUDA_CHECK_ERROR(
            hipMalloc(&finalCoords, sizeof(int2) * numTargets)
        );

        int* finalFacesForCoords;
        CUDA_CHECK_ERROR(
            hipMalloc(&finalFacesForCoords, sizeof(int) * numTargets)
        );

        blockWidth = 32;
        numBlocks = div_ceil(numSamples, blockWidth);

        KERNEL_LAUNCH(
            AA::gather_target_coords,
            numBlocks, blockWidth,
            pCudaBSP,
            finalCoords, finalFacesForCoords,
            coordsForSamples, facesForSamples,
            targetsScanned, targets
        );

        numBlocks = div_ceil(numTargets, blockWidth);

        std::cout << "numBlocks: " << numBlocks << std::endl;

        KERNEL_LAUNCH(
            AA::antialias_coords,
            numBlocks, blockWidth,
            pCudaBSP,
            finalCoords, finalFacesForCoords,
            static_cast<size_t>(numTargets)
        );

        CUDA_CHECK_ERROR(hipDeviceSynchronize());

        CUDA_CHECK_ERROR(hipFree(finalFacesForCoords));
        CUDA_CHECK_ERROR(hipFree(finalCoords));
        CUDA_CHECK_ERROR(hipFree(targetsScanned));
        CUDA_CHECK_ERROR(hipFree(targets));
        CUDA_CHECK_ERROR(hipFree(coordsForSamples));
        CUDA_CHECK_ERROR(hipFree(facesForSamples));

        //const size_t BLOCK_WIDTH = 16;
        //const size_t BLOCK_HEIGHT = 16;

        //size_t numFaces = bsp.get_faces().size();

        //dim3 blockDim(BLOCK_WIDTH, BLOCK_HEIGHT);

        //KERNEL_LAUNCH(
        //    DirectLighting::map_faces_AA,
        //    numFaces, blockDim,
        //    pCudaBSP
        //);

        CUDA_CHECK_ERROR(hipEventRecord(stopEvent));

        CUDA_CHECK_ERROR(hipDeviceSynchronize());

        float time;
        CUDA_CHECK_ERROR(hipEventElapsedTime(&time, startEvent, stopEvent));

        std::cout << "Done! (" << time << " ms)" << std::endl;
    }

    void bounce_lighting(BSP::BSP& bsp, CUDABSP::CUDABSP* pCudaBSP) {
        using Clock = std::chrono::high_resolution_clock;

        auto start = Clock::now();

        auto end = Clock::now();
        std::chrono::milliseconds ms
            = std::chrono::duration_cast<std::chrono::milliseconds>(
                end - start
            );

        std::cout << "Done! (" << ms.count() << " ms)" << std::endl;
    }

    void compute_ambient_lighting(CUDABSP::CUDABSP* pCudaBSP) {
        using Clock = std::chrono::high_resolution_clock;

        auto start = Clock::now();

        const size_t BLOCK_WIDTH = 32;

        size_t numLeaves;

        CUDA_CHECK_ERROR(
            hipMemcpy(
                &numLeaves, &pCudaBSP->numLeaves, sizeof(size_t),
                hipMemcpyDeviceToHost
            )
        );

        KERNEL_LAUNCH(
            AmbientLighting::map_leaves,
            numLeaves, BLOCK_WIDTH,
            pCudaBSP
        );

        CUDA_CHECK_ERROR(hipDeviceSynchronize());

        auto end = Clock::now();
        std::chrono::milliseconds ms
            = std::chrono::duration_cast<std::chrono::milliseconds>(
                end - start
                );

        std::cout << "Done! (" << ms.count() << " ms)" << std::endl;
    }
}
